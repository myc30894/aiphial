#include "hip/hip_runtime.h"
/*
 * File:   main.cpp
 * Author: nickl
 *
 * Created on 28 Декабрь 2009 г., 20:21
 */


#include "nbcudawa.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <cutil.h>

#include "ru_nickl_meanshift_direct_cuda_NativeCudaMSFilter.h"

typedef struct LuvPoint
{
    float x;
    float y;
    float l;
    float u;
    float v;
};

#define GET_PITCHED(TYPE,ARR,PITCH,X,Y) (((TYPE*)((char* )ARR+Y*PITCH))+X)

__device__ LuvPoint GetElement(const LuvPoint* A,int pitch, int x, int y)
{
    return *(((LuvPoint*)((char* )A+y*pitch))+x);
   // return A[y*width+x];
    
}

__device__ void SetElement(LuvPoint* A,int pitch, int x, int y,LuvPoint value )
{
   *(((LuvPoint*)((char* )A+y*pitch))+x)=value;
    //A[y*width+x] = value;
}

__device__ void GetPointsWithin(LuvPoint* result, size_t* resultsize, const LuvPoint* src,int pitch,size_t width,size_t height, int x, int y, int squareRange)
{
    
    int size = 0;

    //printf("squareRange=%d\n",squareRange);

        for (int y0 = y - squareRange; y0 <= y + squareRange; y0++)
        {

            if (y0 >= 0 && y0 < height)
            {

                for (int x0 = x - squareRange; x0 <= x + squareRange; x0++)
                {
                    if (x0 >= 0 && x0 < width)
                    {
                        result[size++]=GetElement(src,pitch,x0, y0);
                    }
                }
            }
        }

    //printf("size=%d\n",size);

     *resultsize=size;
}

__device__ double colorDistance(LuvPoint a, LuvPoint b)
    {
        return (a.l - b.l)*(a.l - b.l) + (a.u - b.u)*(a.u - b.u) + (a.v - b.v)*(a.v - b.v);
    }

__device__ double fullDistance(LuvPoint a, LuvPoint b)
    {
        return (a.l - b.l)*(a.l - b.l) + (a.u - b.u)*(a.u - b.u) + (a.v - b.v)*(a.v - b.v)+ (a.x - b.x)*(a.x - b.x) + (a.y - b.y)*(a.y - b.y);
    }


__global__ void MeanShiftsplit(int a, int b, LuvPoint* srcArray,size_t pitch_src, LuvPoint* resultArray,size_t pitch_res,LuvPoint* tempArray,size_t pitch_temp, size_t h, size_t w, int st, float cs)
{


    int i = a * blockDim.x + threadIdx.x;
    int j = b * blockDim.y + threadIdx.y;

    //__syncthreads();

    if(i < w && j< h)
    {

    //LuvPoint shiftresult={i,j,0,0,0};
    
    LuvPoint shiftingpoint = GetElement(srcArray,pitch_src,i,j);

    LuvPoint shiftresult=shiftingpoint;



    //LuvPoint inSquare[729];


    //LuvPoint* inSquare =GET_PITCHED(LuvPoint,tempArray,pitch_temp,(threadIdx.x*(2*st+1)*(2*st+1)),threadIdx.y);

    //LuvPoint* inSquare = (((LuvPoint*)((char* )tempArray+threadIdx.y*pitch_temp))+threadIdx.x*(2*st+1)*(2*st+1));

    //LuvPoint* inSquare = tempArray +  blockIdx.y * blockDim.y *(2*st+1)*(2*st+1)  +  blockIdx.x * (2*st+1)*(2*st+1);

    LuvPoint* inSquare = tempArray + threadIdx.x * blockDim.y *(2*st+1)*(2*st+1)  + threadIdx.y * (2*st+1)*(2*st+1);

    //printf("inSquare=%d\n",inSquare);

    size_t inSquareSize = 0;

  //  printf("i=%d, j=%d, threadIdx.x=%d, threadIdx.y=%d, inSquare=%d\n",i, j,threadIdx.x,threadIdx.y,long(inSquare));

//    printf("i=%d, j=%d,\n",i, j);


//    if(i==0 && j == 32)
//    {
//
    


        LuvPoint prevposition;

        for (int s = 0; s < 10; s++)
        {

            prevposition = shiftresult;

         //   printf("sr = l=%f, u=%f, v=%f, x=%f, y=%f,\n",shiftresult.l,shiftresult.u,shiftresult.v, shiftresult.x, shiftresult.y);
         //   printf("x=%d, y=%d,\n",int(shiftresult.x + 0.5), int(shiftresult.y + 0.5));
            
            GetPointsWithin(inSquare, &inSquareSize, srcArray, pitch_src, w, h, int(shiftresult.x + 0.5), int(shiftresult.y + 0.5), st);

          //  printf("inSquareSizee=%d\n",inSquareSize);

            int count = 0;

            shiftresult.l = 0.;
            shiftresult.u = 0.;
            shiftresult.v = 0.;
            shiftresult.x = 0.;
            shiftresult.y = 0.;

            for (int k = 0; k < inSquareSize; k++)
            {
                LuvPoint point = inSquare[k];

//                printf("colorDistance(point, prevposition)=%f, cs * cs = %f\n",colorDistance(point, prevposition), cs * cs);
//                printf("point = l=%f, u=%f, v=%f, x=%f, y=%f,\n",point.l,point.u,point.v, point.x, point.y);

                if (colorDistance(point, prevposition) < cs * cs)
                {
                    shiftresult.l += point.l;
                    shiftresult.u += point.u;
                    shiftresult.v += point.v;
                    shiftresult.x += point.x;
                    shiftresult.y += point.y;
                    count++;
                }
            }
          //  printf("count=%d\n",count);
            

            shiftresult.l /= count;
            shiftresult.u /= count;
            shiftresult.v /= count;
            shiftresult.x /= count;
            shiftresult.y /= count;

           //  printf("l=%f, u=%f, v=%f, x=%f, y=%f,\n",shiftresult.l,shiftresult.u,shiftresult.v, int(shiftresult.x + 0.5), int(shiftresult.y + 0.5));

            if(colorDistance(prevposition, shiftresult)<3)
                break;

        }//while(fullDistance(prevposition, shiftresult)>10);


        shiftresult.x = i;
        shiftresult.y = j;

//}
    SetElement(resultArray,pitch_res,i,j,shiftresult );

    }

    //__syncthreads();


}


extern "C" void processsplit(double* srcArray, double* resultArray, size_t h, size_t w, int st, float cs)
{

    //st = 0;
    //h=300;
   // w=200;

    //fprintf(stderr,"using processsplit\n");



    LuvPoint* srcCuda;
    size_t pitch_src = w * sizeof (LuvPoint);
    CUDA_SAFE_CALL(hipMallocPitch((void**) & srcCuda, &pitch_src, w * sizeof (LuvPoint), h));
    //CUDA_SAFE_CALL(hipMalloc((void**) & srcCuda, w * sizeof (LuvPoint) * h));


    LuvPoint* resCuda;
    size_t pitch_res = w * sizeof (LuvPoint);
    CUDA_SAFE_CALL(hipMallocPitch((void**) & resCuda, &pitch_res, w * sizeof (LuvPoint), h));
    //CUDA_SAFE_CALL(hipMalloc((void**) & resCuda, w * sizeof (LuvPoint) * h));


   LuvPoint* src = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));
    for (int i = 0; i < w * h; i++)
    {
        src[i].l = srcArray[i * 3 + 0];
        src[i].u = srcArray[i * 3 + 1];
        src[i].v = srcArray[i * 3 + 2];
        src[i].x = i%w;
        src[i].y = i/w;
    }

    LuvPoint* dest = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));

    //memcpy(dest,src,w*h*sizeof (Luv));



    CUDA_SAFE_CALL(hipMemcpy2D(srcCuda, pitch_src, src, w * sizeof (LuvPoint), w * sizeof (LuvPoint), h, hipMemcpyHostToDevice));

 //   CUDA_SAFE_CALL(hipMemcpy2D(resCuda, pitch_res, srcCuda, pitch_src, w * sizeof (LuvPoint), h, hipMemcpyDeviceToDevice));
//

    dim3 dimBlock(16,16);
    int amax = (w + dimBlock.x -1)/dimBlock.x;
    int bmax = (h+dimBlock.y -1)/dimBlock.y;
    dim3 dimGrid(1,1);

    LuvPoint* tempSqDataArray;
    size_t pitch_tempSqData;

    //CUDA_SAFE_CALL(hipMallocPitch((void**) & tempSqDataArray, &pitch_tempSqData, dimBlock.x *(2*st+1)*(2*st+1) * sizeof (LuvPoint), dimBlock.y));

    size_t blockLength = (2*st+1)*(2*st+1) * sizeof (LuvPoint);

    CUDA_SAFE_CALL(hipMalloc((void**) & tempSqDataArray, dimBlock.x * dimBlock.y *blockLength ));


       for (int b = 0; b < bmax; b++)
        {

    for (int a = 0; a < amax; a++)
    {
     

        // MeanShift<<<dimGrid, dimBlock>>>(a,b,(((LuvPoint*)((char* )srcCuda+b*pitch_src))+a),pitch_src, (((LuvPoint*)((char* )resCuda+b*pitch_res))+a),pitch_res,tempSqDataArray,pitch_tempSqData,h, w, st, cs);
        MeanShiftsplit<<<dimGrid, dimBlock>>>(a,b,srcCuda,pitch_src, resCuda,pitch_res,tempSqDataArray,pitch_tempSqData,h, w, st, cs);

           //hipDeviceSynchronize();


         CUDA_SAFE_CALL(hipGetLastError());

        }
    }



    CUDA_SAFE_CALL(hipFree(tempSqDataArray));

    CUDA_SAFE_CALL(hipMemcpy2D(dest,w * sizeof (LuvPoint), resCuda, pitch_res, w * sizeof (LuvPoint), h, hipMemcpyDeviceToHost));


    for (int i = 0; i < w * h; i++)
    {
        resultArray[i * 3 + 0] = dest[i].l;
        resultArray[i * 3 + 1] = dest[i].u;
        resultArray[i * 3 + 2] = dest[i].v;
    }



    free(dest);
    free(src);
    CUDA_SAFE_CALL(hipFree(srcCuda));
    CUDA_SAFE_CALL(hipFree(resCuda));

}


__global__ void MeanShiftstack( LuvPoint* srcArray,size_t pitch_src, LuvPoint* resultArray,size_t pitch_res, size_t h, size_t w, int st, float cs)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y  * blockDim.y + threadIdx.y;

    //__syncthreads();

    if(i < w && j< h)
    {

    //LuvPoint shiftresult={i,j,0,0,0};

    LuvPoint shiftingpoint = GetElement(srcArray,pitch_src,i,j);

    LuvPoint shiftresult=shiftingpoint;



    LuvPoint inSquare[729];


    size_t inSquareSize = 0;

        LuvPoint prevposition;

        for (int s = 0; s < 10; s++)
        {

            prevposition = shiftresult;

            GetPointsWithin(inSquare, &inSquareSize, srcArray, pitch_src, w, h, int(shiftresult.x + 0.5), int(shiftresult.y + 0.5), st);

            int count = 0;

            shiftresult.l = 0.;
            shiftresult.u = 0.;
            shiftresult.v = 0.;
            shiftresult.x = 0.;
            shiftresult.y = 0.;

            for (int k = 0; k < inSquareSize; k++)
            {
                LuvPoint point = inSquare[k];

                if (colorDistance(point, prevposition) < cs * cs)
                {
                    shiftresult.l += point.l;
                    shiftresult.u += point.u;
                    shiftresult.v += point.v;
                    shiftresult.x += point.x;
                    shiftresult.y += point.y;
                    count++;
                }
            }


            shiftresult.l /= count;
            shiftresult.u /= count;
            shiftresult.v /= count;
            shiftresult.x /= count;
            shiftresult.y /= count;

            if(colorDistance(prevposition, shiftresult)<3)
                break;

        }

        shiftresult.x = i;
        shiftresult.y = j;


    SetElement(resultArray,pitch_res,i,j,shiftresult );

    }

    //__syncthreads();


}


extern "C" void processstack(double* srcArray, double* resultArray, size_t h, size_t w, int st, float cs)
{

    //st = 0;
    //h=300;
   // w=200;

    //fprintf(stderr,"using processstack\n");

     if(st>13)
         fprintf(stderr,"spatial side cannot be more than 13, setted value is %d\n", st);
        return;
    
    LuvPoint* srcCuda;
    size_t pitch_src = w * sizeof (LuvPoint);
    CUDA_SAFE_CALL(hipMallocPitch((void**) & srcCuda, &pitch_src, w * sizeof (LuvPoint), h));
    //CUDA_SAFE_CALL(hipMalloc((void**) & srcCuda, w * sizeof (LuvPoint) * h));


    LuvPoint* resCuda;
    size_t pitch_res = w * sizeof (LuvPoint);
    CUDA_SAFE_CALL(hipMallocPitch((void**) & resCuda, &pitch_res, w * sizeof (LuvPoint), h));
    //CUDA_SAFE_CALL(hipMalloc((void**) & resCuda, w * sizeof (LuvPoint) * h));


   LuvPoint* src = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));
    for (int i = 0; i < w * h; i++)
    {
        src[i].l = srcArray[i * 3 + 0];
        src[i].u = srcArray[i * 3 + 1];
        src[i].v = srcArray[i * 3 + 2];
        src[i].x = i%w;
        src[i].y = i/w;
    }

    LuvPoint* dest = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));

    //memcpy(dest,src,w*h*sizeof (Luv));



    CUDA_SAFE_CALL(hipMemcpy2D(srcCuda, pitch_src, src, w * sizeof (LuvPoint), w * sizeof (LuvPoint), h, hipMemcpyHostToDevice));

 //   CUDA_SAFE_CALL(hipMemcpy2D(resCuda, pitch_res, srcCuda, pitch_src, w * sizeof (LuvPoint), h, hipMemcpyDeviceToDevice));
//

    dim3 dimBlock(16,16);
    dim3 dimGrid((w + dimBlock.x -1)/dimBlock.x,(h+dimBlock.y -1)/dimBlock.y);

    LuvPoint* tempSqDataArray;
    size_t pitch_tempSqData;

    //CUDA_SAFE_CALL(hipMallocPitch((void**) & tempSqDataArray, &pitch_tempSqData, dimBlock.x *(2*st+1)*(2*st+1) * sizeof (LuvPoint), dimBlock.y));

    size_t blockLength = (2*st+1)*(2*st+1) * sizeof (LuvPoint);


    MeanShiftstack<<<dimGrid, dimBlock>>>(srcCuda,pitch_src, resCuda,pitch_res,h, w, st, cs);
    //hipDeviceSynchronize();


    CUDA_SAFE_CALL(hipGetLastError());


    CUDA_SAFE_CALL(hipMemcpy2D(dest,w * sizeof (LuvPoint), resCuda, pitch_res, w * sizeof (LuvPoint), h, hipMemcpyDeviceToHost));


    for (int i = 0; i < w * h; i++)
    {
        resultArray[i * 3 + 0] = dest[i].l;
        resultArray[i * 3 + 1] = dest[i].u;
        resultArray[i * 3 + 2] = dest[i].v;
    }



    free(dest);
    free(src);
    CUDA_SAFE_CALL(hipFree(srcCuda));
    CUDA_SAFE_CALL(hipFree(resCuda));

}



//
//extern "C" void processfull(double* srcArray, double* resultArray, size_t h, size_t w, int st, float cs)
//{
//
//    //st = 0;
//    //h=300;
//   // w=200;
//
//    LuvPoint* srcCuda;
//    size_t pitch_src = w * sizeof (LuvPoint);
//    CUDA_SAFE_CALL(hipMallocPitch((void**) & srcCuda, &pitch_src, w * sizeof (LuvPoint), h));
//    //CUDA_SAFE_CALL(hipMalloc((void**) & srcCuda, w * sizeof (LuvPoint) * h));
//
//
//    LuvPoint* resCuda;
//    size_t pitch_res = w * sizeof (LuvPoint);
//    CUDA_SAFE_CALL(hipMallocPitch((void**) & resCuda, &pitch_res, w * sizeof (LuvPoint), h));
//    //CUDA_SAFE_CALL(hipMalloc((void**) & resCuda, w * sizeof (LuvPoint) * h));
//
//
//   LuvPoint* src = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));
//    for (int i = 0; i < w * h; i++)
//    {
//        src[i].l = srcArray[i * 3 + 0];
//        src[i].u = srcArray[i * 3 + 1];
//        src[i].v = srcArray[i * 3 + 2];
//        src[i].x = i%w;
//        src[i].y = i/w;
//    }
//
//    LuvPoint* dest = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));
//
//    //memcpy(dest,src,w*h*sizeof (Luv));
//
//
//
//    CUDA_SAFE_CALL(hipMemcpy2D(srcCuda, pitch_src, src, w * sizeof (LuvPoint), w * sizeof (LuvPoint), h, hipMemcpyHostToDevice));
//
// //   CUDA_SAFE_CALL(hipMemcpy2D(resCuda, pitch_res, srcCuda, pitch_src, w * sizeof (LuvPoint), h, hipMemcpyDeviceToDevice));
////
//
//    dim3 dimBlock(16,16);
//    dim3 dimGrid((w + dimBlock.x -1)/dimBlock.x,(h+dimBlock.y -1)/dimBlock.y);
//
//    LuvPoint* tempSqDataArray;
//    size_t pitch_tempSqData;
//
//    //CUDA_SAFE_CALL(hipMallocPitch((void**) & tempSqDataArray, &pitch_tempSqData, dimBlock.x *(2*st+1)*(2*st+1) * sizeof (LuvPoint), dimBlock.y));
//
//    size_t blockLength = (2*st+1)*(2*st+1) * sizeof (LuvPoint);
//
//    CUDA_SAFE_CALL(hipMalloc((void**) & tempSqDataArray, dimBlock.x * dimBlock.y *blockLength ));
//
//    LuvPoint* tempfiller = (LuvPoint*) malloc(dimBlock.x * dimBlock.y *blockLength);
//
//    for (int i = 0; i < dimBlock.x * dimBlock.y * (2*st+1)*(2*st+1); i++)
//    {
//
//        tempfiller[i].l = 0.;
//        tempfiller[i].u = 0.;
//        tempfiller[i].v = 0.;
//        tempfiller[i].x = 0.;
//        tempfiller[i].y = 0.;
//    }
//
//    CUDA_SAFE_CALL(hipMemcpy(tempSqDataArray,tempfiller, dimBlock.x * dimBlock.y *blockLength, hipMemcpyHostToDevice));
//
//
//
//    printf("tempSqDataArraystart=%d, blockLength=%d\n",long(tempSqDataArray), blockLength);
//
//    printf("a=%d, b=%d\n",dimGrid.x,dimGrid.y);
//
//    printf("h=%d, w=%d, st=%d, cs=%f,\n",h, w, st, cs);
//
//    MeanShift<<<dimGrid, dimBlock>>>(srcCuda,pitch_src, resCuda,pitch_res,tempSqDataArray,pitch_tempSqData,h, w, st, cs);
//    //hipDeviceSynchronize();
//
//
//    CUDA_SAFE_CALL(hipGetLastError());
//
//
//    CUDA_SAFE_CALL(hipMemcpy(tempfiller, tempSqDataArray, dimBlock.x * dimBlock.y *blockLength, hipMemcpyDeviceToHost));
//
//    for (int i = 0; i < dimBlock.x * dimBlock.y * (2*st+1)*(2*st+1); i++)
//    {
//
//        LuvPoint p = tempfiller[i];
//
//        //printf("%f %f\n",p.x,p.y);
//    }
//
//    free(tempfiller);
//    CUDA_SAFE_CALL(hipFree(tempSqDataArray));
//
//    CUDA_SAFE_CALL(hipMemcpy2D(dest,w * sizeof (LuvPoint), resCuda, pitch_res, w * sizeof (LuvPoint), h, hipMemcpyDeviceToHost));
//
//
//    for (int i = 0; i < w * h; i++)
//    {
//        resultArray[i * 3 + 0] = dest[i].l;
//        resultArray[i * 3 + 1] = dest[i].u;
//        resultArray[i * 3 + 2] = dest[i].v;
//    }
//
//
//
//    free(dest);
//    free(src);
//    CUDA_SAFE_CALL(hipFree(srcCuda));
//    CUDA_SAFE_CALL(hipFree(resCuda));
//
//}
//

/*
 */

// <editor-fold defaultstate="collapsed" desc="deformed process1">

__global__ void deformed(LuvPoint* A, LuvPoint* B,
                         LuvPoint* C, LuvPoint* srcArray, LuvPoint* resultArray, LuvPoint* tempArray, size_t h, size_t w, int st, double cs)
{



    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= w || j >= h)
        return;


    LuvPoint shiftresult = {i, j, 100, 100, 100};

    LuvPoint shiftingpoint = GetElement(srcArray, w, i, j);

    SetElement(resultArray, w, i, j, shiftresult);


    (*(C + j * w + i)).l = (*(A + j * w + i)).l + (*(B + j * w + i)).l;
    (*(C + j * w + i)).u = (*(A + j * w + i)).u + (*(B + j * w + i)).u;
    (*(C + j * w + i)).v = (*(A + j * w + i)).v + (*(B + j * w + i)).v;




}

extern "C" void process1(double* srcArray, double* resultArray, size_t h, size_t w, int st, double cs)
{

    // h = 300;
    // w = 300;

    LuvPoint* srcCuda;
    //size_t pitch_src;// = w * sizeof (LuvPoint);
    //CUDA_SAFE_CALL(hipMallocPitch((void**) & srcCuda, &pitch_src, w * sizeof (LuvPoint), h));
    CUDA_SAFE_CALL(hipMalloc((void**) & srcCuda, w * sizeof (LuvPoint) * h));


    LuvPoint* resCuda;
    //size_t pitch_res;// = w * sizeof (LuvPoint);
    //CUDA_SAFE_CALL(hipMallocPitch((void**) & resCuda, &pitch_res, w * sizeof (LuvPoint), h));
    CUDA_SAFE_CALL(hipMalloc((void**) & resCuda, w * sizeof (LuvPoint) * h));


    LuvPoint* src = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));
    for (int i = 0; i < w * h; i++)
    {
        src[i].l = srcArray[i * 3 + 0];
        src[i].u = srcArray[i * 3 + 1];
        src[i].v = srcArray[i * 3 + 2];
        src[i].x = i / w;
        src[i].y = i % w;
    }

    LuvPoint* dest = (LuvPoint*) malloc(w * h * sizeof (LuvPoint));

    //memcpy(dest,src,w*h*sizeof (Luv));



    CUDA_SAFE_CALL(hipMemcpy(srcCuda, src, w * sizeof (LuvPoint) * h, hipMemcpyHostToDevice));

    //   CUDA_SAFE_CALL(hipMemcpy2D(resCuda, pitch_res, srcCuda, pitch_src, w * sizeof (LuvPoint), h, hipMemcpyDeviceToDevice));
    //
    LuvPoint* tempSqDataArray;
    //size_t pitch_tempSqData;
    CUDA_SAFE_CALL(hipMalloc((void**) & tempSqDataArray, w * sizeof (LuvPoint) * st * st * h));

    printf("h=%d, w=%d, st=%d, cs=%f,\n", h, w, st, cs);


    //   dim3 dimBlock(16, 16);
    //   dim3 dimGrid((w + dimBlock.x – 1) / dimBlock.x,(h + dimBlock.y – 1) / dimBlock.y);

    //    dim3 dimBlock(8,8);
    //    dim3 dimGrid((w + dimBlock.x -1)/dimBlock.x,(h+dimBlock.y -1)/dimBlock.y);
    //
    //    printf("a=%d, b=%d\n",(w + dimBlock.x -1)/dimBlock.x,(h+dimBlock.y -1)/dimBlock.y);

    //MeanShift<<<dimGrid, dimBlock>>>(srcCuda,pitch_src, resCuda,pitch_res,tempSqDataArray,pitch_tempSqData,h, w, st, cs);



    // -------------------------------------------------

    printf("h=%d, w=%d, st=%d, cs=%f,\n", h, w, st, cs);

    int size = h * w * sizeof (LuvPoint);
    // Allocate input vectors h_A and h_B in host memory
    LuvPoint* h_A = (LuvPoint*) malloc(size);
    LuvPoint* h_B = (LuvPoint*) malloc(size);
    LuvPoint* h_C = (LuvPoint*) malloc(size);
    // Allocate vectors in device memory
    LuvPoint* d_A;
    CUDA_SAFE_CALL(hipMalloc((void**) & d_A, size));
    LuvPoint* d_B;
    CUDA_SAFE_CALL(hipMalloc((void**) & d_B, size));
    LuvPoint* d_C;
    CUDA_SAFE_CALL(hipMalloc((void**) & d_C, size));

    for (int i = 0; i < h * w; i++)
    {
        h_A[i].l = i;
        h_A[i].u = i;
        h_A[i].v = i;
        h_B[i].l = i;
        h_B[i].u = i;
        h_B[i].v = i;

    }

    // Copy vectors from host memory to device memory
    CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    // Invoke kernel

    dim3 dimBlock1(16, 16);
    dim3 dimGrid1((w + dimBlock1.x - 1) / dimBlock1.x, (h + dimBlock1.y - 1) / dimBlock1.y);

    printf("a=%d, b=%d\n", (w + dimBlock1.x - 1) / dimBlock1.x, (h + dimBlock1.y - 1) / dimBlock1.y);

    deformed <<<dimGrid1, dimBlock1>>>(d_A, d_B, d_C, srcCuda, resCuda, tempSqDataArray, h, w, st, cs);

    //deformed<<<dimGrid1, dimBlock1>>>(d_A, d_B, d_C,0,0, 0,0,0,0,h, w, st, cs);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipGetLastError());


    for (int i = 0; i < w * h && i < 10; i++)
    {
        printf("l=%0.2f u=%0.2f v=%0.2f, ", h_A[i].l, h_A[i].u, h_A[i].v);
    }
    printf("\n");
    for (int i = 0; i < w * h && i < 10; i++)
    {
        printf("l=%0.2f u=%0.2f v=%0.2f, ", h_B[i].l, h_B[i].u, h_B[i].v);
    }
    printf("\n");
    for (int i = 0; i < w * h && i < 10; i++)
    {
        printf("l=%0.2f u=%0.2f v=%0.2f, ", h_C[i].l, h_C[i].u, h_C[i].v);
    }
    printf("\n");
    // Free device memory
    CUDA_SAFE_CALL(hipFree(d_A));

    CUDA_SAFE_CALL(hipFree(d_B));
    CUDA_SAFE_CALL(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(h_C);

    // ----------------------------------------------------------


    CUDA_SAFE_CALL(hipGetLastError());

    CUDA_SAFE_CALL(hipFree(tempSqDataArray));

    CUDA_SAFE_CALL(hipMemcpy(dest, resCuda, w * sizeof (LuvPoint) * h, hipMemcpyDeviceToHost));


    for (int i = 0; i < w * h; i++)
    {
        resultArray[i * 3 + 0] = dest[i].l;
        resultArray[i * 3 + 1] = dest[i].u;
        resultArray[i * 3 + 2] = dest[i].v;
    }

    free(dest);
    free(src);
    CUDA_SAFE_CALL(hipFree(srcCuda));
    CUDA_SAFE_CALL(hipFree(resCuda));



}
// </editor-fold>



JNIEXPORT void JNICALL Java_ru_nickl_meanshift_direct_cuda_NativeCudaMSFilter_doNativefilter
(JNIEnv *env, jobject obj, jdoubleArray resultJarray, jdoubleArray srcJarray, jint h, jint w, jshort st, jdouble cs, jint mode)
{
    jdouble* srcarray = (env)->GetDoubleArrayElements(srcJarray, NULL);

    //(*env)->SetDoubleArrayRegion(env,resultJarray,0,h*w*3,srcarray);

   jdouble* resultarray = (env)->GetDoubleArrayElements(resultJarray, NULL);


    //processstack(srcarray, resultarray, h, w, st, cs);

   switch(mode){
   case 0: processsplit(srcarray, resultarray, h, w, st, cs); break;
   case 1: processstack(srcarray, resultarray, h, w, st, cs); break;
   }


    (env)->ReleaseDoubleArrayElements(resultJarray, resultarray, 0);
    (env)->ReleaseDoubleArrayElements(srcJarray, srcarray, 0);

}



// Host code
